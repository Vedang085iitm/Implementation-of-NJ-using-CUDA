#include <bits/stdc++.h>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

__global__ void rowsum(int *d_matrix, int *d_sums, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long i = block_Idx * block_Capacity + thread_Idx;

    if (i < n)
    {
        d_sums[i] = 0; // Initialize the sum to 0
        for (int j = 0; j < n; ++j)
        {
            atomicAdd(&d_sums[i], d_matrix[i * n + j]);
        }
    }
}

__global__ void neighborJoiningMatrix(int *d_matrix, int *d_rowSums, int *d_njMatrix, int n)
{
    long block_Idx = blockIdx.x + (gridDim.x) * blockIdx.y + (gridDim.y * gridDim.x) * blockIdx.z;
    long thread_Idx = threadIdx.x + (blockDim.x) * threadIdx.y + (blockDim.y * blockDim.x) * threadIdx.z;
    long block_Capacity = blockDim.x * blockDim.y * blockDim.z;
    long arr_Idx = block_Idx * block_Capacity + thread_Idx;

    if (arr_Idx < n * n)
    {
        int i = arr_Idx / n;
        int j = arr_Idx % n;

        if (i == j)
        {
            d_njMatrix[arr_Idx] = 0;
        }
        else
        {
            d_njMatrix[arr_Idx] = (n - 2) * d_matrix[arr_Idx] - d_rowSums[i] - d_rowSums[j];
        }
    }
}

int main()
{
    int n;
    cout << "Enter the size of the matrix: " << endl;
    cin >> n;
    cout << "Enter the matrix: " << endl;
    int *matrix = new int[n * n];
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cin >> matrix[i * n + j];
        }
    }
    int *sums = new int[n];
    int *d_matrix;
    int *d_sums;
    hipMalloc(&d_matrix, n * n * sizeof(int));
    hipMalloc(&d_sums, n * sizeof(int));
    hipMemcpy(d_matrix, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(512);
    dim3 numBlocks((n + blockSize.x - 1) / blockSize.x);

    rowsum<<<numBlocks, blockSize>>>(d_matrix, d_sums, n);
    hipDeviceSynchronize();
    cout << "Row sums: " << endl;
    hipMemcpy(sums, d_sums, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i)
    {
        cout << sums[i] << " ";
    }
    cout << endl;

    int *njMatrix = new int[n * n];
    int *d_njMatrix;
    hipMalloc(&d_njMatrix, n * n * sizeof(int));
    neighborJoiningMatrix<<<numBlocks, blockSize>>>(d_matrix, d_sums, d_njMatrix, n);
    hipDeviceSynchronize();
    hipMemcpy(njMatrix, d_njMatrix, n * n * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Neighbor joining matrix: " << endl;
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            cout << njMatrix[i * n + j] << " ";
        }
        cout << endl;
    }

    delete[] matrix;
    delete[] sums;
    hipFree(d_matrix);
    hipFree(d_sums);
    hipFree(d_njMatrix);
    
    return 0;
}