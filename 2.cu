#include <vector>
#include <iostream>
#include <numeric>
#include <chrono>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <fstream>
# define ll long long

using namespace std;

using std::cin;
using std::cout;


__global__ void getsum(int * rowsums , int * matrix , int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n*n){    
        int row = idx/n;
        atomicAdd(&rowsums[row] , matrix[idx]);
    }
}


__global__ void create_nj(int *matrix, int *rowsums, int *njMat, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n * n) {
        int i = idx / n;
        int j = idx % n;
        int value = matrix[idx];  // Fetch matrix element once

        if (i == j) {
            njMat[idx] = 0;
        } else {
            int nj_value = (n - 2) * value - rowsums[i] - rowsums[j];
            njMat[idx] = nj_value;
        }
    }
}


__global__ void findMin(int *njMat, int *result, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Initialize the result with the maximum integer value
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *result = INT_MAX;
    
    __syncthreads();

    // Each thread finds the minimum value it has access to
    int minValue = (idx < n * n) ? njMat[idx] : INT_MAX;
    
    // Reduction to find the minimum value among all elements
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        minValue = min(minValue, __shfl_down_sync(0xFFFFFFFF, minValue, stride));
    }

    // The first thread in each block updates the result
    if (threadIdx.x == 0) {
        atomicMin(result, minValue);
    }
}

__global__ void getMinidx(int * njMat , int * min_idx ,  int minVal , int n){
    int idx = blockDim.x *blockIdx.x + threadIdx.x;
    if(idx < n*n){
        if(njMat[idx] == minVal){
            min_idx[0] = idx;
        }
    }
}

__global__ void makeNew(int *old_matrix, int *new_matrix, int i, int j, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n * n) {
        int row = idx / n;
        int col = idx % n;
        int new_row, new_col;

        if (row < i) {
            new_row = row;
        } else if (row > j) {
            new_row = row - 1;
        } else {
            return; // Skip the elements in rows i and j
        }

        if (col < i) {
            new_col = col;
        } else if (col > j) {
            new_col = col - 1;
        } else {
            return; // Skip the elements in columns i and j
        }

        new_matrix[new_row * (n - 1) + new_col] = old_matrix[idx];
    }
}


__global__ void makeMerge(int *new_matrix, int *old_matrix, int *map, int i, int j, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Only calculate for valid indices
    if (idx < n - 1) {
        int actual_idx = map[idx + 1]; // Offset idx by 1 since idx == 0 is handled differently
        int old_matrix_i_actual_idx = n * i + actual_idx;
        int old_matrix_j_actual_idx = n * j + actual_idx;
        int old_matrix_i_j = old_matrix[n * i + j];

        // Handle first element of new_matrix separately
        if (idx == 0) {
            new_matrix[0] = 0;
        } else {
            // Calculate new_matrix values
            int sum = old_matrix[old_matrix_i_actual_idx] + old_matrix[old_matrix_j_actual_idx] - old_matrix_i_j;
            int new_val = sum / 2;
            new_matrix[idx] = new_val;
            new_matrix[(n - 1) * idx] = new_val;
        }
    }
}


pair<int, int> calculateLimbLengths(int * matrix, int i, int j, int delta , int n) {
    int limbLengthI = (matrix[n*i +j] + delta) / 2;
    int limbLengthJ = (matrix[n*i +j] - delta) / 2;
    return make_pair(limbLengthI, limbLengthJ);
}

int main(int argc, char **argv){
    cout << "Print -" << endl;
    int n;
    cin >> n;
    cout << "Print- " << endl;
    int *matrix = new int[n * n];
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            cin >> matrix[i * n + j];
        }
    }

    ofstream outfile("cuda.out");

    

    int top = n;
    vector<int> prev(n , 0);
    vector<int> next(n , 0);
    vector<vector<int>> tree(n);
    vector<vector<int>> edgeWeights(1e4 , vector<int> (1e4 , 0));
    int * new_mat = new int[n*n];
    int * rowsums = new int[n];
    int * njMatrix = new int[n * n];
    int * d_old_mat;
    int * d_new_mat;
    int * d_rowsums;
    int * d_njMatrix;

    int minVal , minIdx , minIdxI , minIdxJ , delta;
    minVal = 1e9;
    int * d_minVal , * d_minIdx;
    
    hipMalloc(&d_njMatrix, n * n * sizeof(int));
    hipMalloc(&d_old_mat, n * n * sizeof(int));
    hipMalloc(&d_new_mat, n * n * sizeof(int));
    hipMalloc(&d_rowsums, n * sizeof(int));
    hipMalloc(&d_minVal, sizeof(int));
    hipMalloc(&d_minIdx , sizeof(int));

    hipMemcpy(d_minVal , &minVal , sizeof(int) , hipMemcpyHostToDevice);

    hipMemcpy(d_old_mat, matrix, n * n * sizeof(int), hipMemcpyHostToDevice);

    for(int i = 0 ; i < n ; i ++) prev[i] = i;

    int num_blocks;
    auto start = chrono::high_resolution_clock::now();

    while(n - 2){

        // get row sums
        num_blocks = (n*n + 1023) / 1024;
        hipMemset(d_rowsums , 0 , n*sizeof(int));

        getsum<<< num_blocks , 1024 >>> (d_rowsums , d_old_mat , n);
        hipDeviceSynchronize();
        hipMemcpy(rowsums , d_rowsums , n*sizeof(int) , hipMemcpyDeviceToHost);

        // create nj matrix
        create_nj <<< num_blocks , 1024 >>> (d_old_mat , d_rowsums , d_njMatrix , n);
        hipDeviceSynchronize();
        hipMemcpy(njMatrix , d_njMatrix , n*n*sizeof(int) , hipMemcpyDeviceToHost);

        //find Minval
        minVal = 1e5;
        hipMemcpy(d_minVal , &minVal , sizeof(int) , hipMemcpyHostToDevice);
        findMin <<< num_blocks , 1024 >>> (d_njMatrix , d_minVal , n);
        hipDeviceSynchronize();
        hipMemcpy(&minVal , d_minVal , sizeof(int) , hipMemcpyDeviceToHost);

        //find MinIndices
        getMinidx <<< num_blocks , 1024 >>> (d_njMatrix , d_minIdx , minVal , n);
        hipDeviceSynchronize();
        hipMemcpy(&minIdx , d_minIdx , sizeof(int) , hipMemcpyDeviceToHost);
        minIdxI = minIdx / n;
        minIdxJ = minIdx % n;
        delta = (rowsums[minIdxI] - rowsums[minIdxJ]) / (n-2);
        cout << "Min Indices - " << minIdxI << " " << minIdxJ << endl;

        // make newMatrix
        makeNew <<< num_blocks , 1024 >>> (d_old_mat , d_new_mat , minIdxI , minIdxJ , n);
        hipDeviceSynchronize();

        hipMemcpy(new_mat , d_new_mat , (n-1)*(n-1)*sizeof(int) ,  hipMemcpyDeviceToHost);
        
        new_mat[0] = 0;
        ll ct = 0;
        for(ll m = 1 ; m < n - 1 ; m++){
            while(ct==minIdxI || ct==minIdxJ) ct++;
            new_mat[m] = (matrix[minIdxI * n + ct] + matrix[minIdxJ * n + ct] - matrix[minIdxI *n + minIdxJ]) / 2;
            new_mat[m*(n-1)] = new_mat[m];
            ct++;  
        }

        tree.push_back(vector<int>());
        
        pair<int,int> pr = calculateLimbLengths(matrix , minIdxI , minIdxJ , delta , n);

        edgeWeights[prev[minIdxI]][top] = pr.first;
        edgeWeights[prev[minIdxJ]][top] = pr.second;
        edgeWeights[top][prev[minIdxI]] = pr.first;
        edgeWeights[top][prev[minIdxJ]] = pr.second;
        tree[top].push_back(prev[minIdxI]);
        tree[top].push_back(prev[minIdxJ]);
        tree[prev[minIdxI]].push_back(top);
        tree[prev[minIdxJ]].push_back(top);

        next[0] = top;
        ll cnt = 1;
        for(ll i = 0 ; i < n ; i++){
            if(i!=minIdxI && i!=minIdxJ) next[cnt++] = prev[i];
        }
        cout << "OldMat" << endl;
        for(int i = 0 ; i < n ; i ++){
            for(int j = 0 ; j < n ; j ++){
                cout << matrix[n*i + j] << " "; 
            }
            cout << endl;
        }
        cout << "New Mat" << endl;
        for(int i = 0 ; i < n -1 ; i++){
            for(int j = 0 ; j < n - 1 ; j++){
                cout << new_mat[i*(n-1) + j] << " ";
            }
            cout << endl;
        }
        prev = next;
        matrix = new_mat;
        hipMemcpy(d_old_mat , matrix , (n-1)*(n-1)*sizeof(int) , hipMemcpyHostToDevice);
        cout << "After copy " << endl;
        for(int i = 0 ; i < n -1 ; i++){
            for(int j = 0 ; j < n - 1 ; j++){
                cout << matrix[i*(n-1) + j] << " ";
            }
            cout << endl;
        }
        top++;
        n--;
    }
    tree[next[0]].push_back(next[1]);
    tree[next[1]].push_back(next[0]);
    edgeWeights[next[0]][next[1]] = new_mat[1];
    edgeWeights[next[1]][next[0]] = new_mat[1];

    auto stop = chrono::high_resolution_clock::now();
    
    auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);

    for(int i = 0 ; i < top ; i++){
        cout << i << " - ";
        for(auto v : tree[i]){
            cout << v << " ";
        }
        cout << endl;
    }

    for(int i = 0 ; i < top ; i++){
        for(int j = 0 ; j < top ; j++){
            cout << edgeWeights[i][j] << " ";
        }
        cout << endl;
    }

    cout << "Time taken by function: " << duration.count() << " microseconds" << endl;
}